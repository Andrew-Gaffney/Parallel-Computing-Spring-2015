#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MAX_RAND 100001
#define WARP_SIZE 32

int size;
int nThreads;


/*Function to round up number of threads to nearest power of 2,
so that algorithm works(kind of a cheap workaround, I know, 
and also terrible for performance)
*/
int NearestPowerOf2 (int n) {
	
	if (!n) return n; 
  
	int x = 1;
  	while(x < n)
    {
      x <<= 1;
    }
  	return x;
}

//Simple function to get user input, set up thread count and fill random array
int* setup(char n[]){
	int i;
	int *rand_num;
	time_t t;
	
	//get size of array
	sscanf(n, "%d", &size);
	
	nThreads = NearestPowerOf2(size);
	
	srand((unsigned) time(&t));
	
	
	rand_num = (int*)malloc(size * sizeof(int));

    
	for(i = 0; i < size; i++){
		rand_num[i] = random() % MAX_RAND;
		//printf("%d\n", i);
	}
	
	return rand_num;	
	
}

/*
After kernel execution, max of input array will be somewhere within index 0
and 31 of array. Kernel is modified so that all threads in each warp are executing the same branch at the same time so as to avoid branch divergence.
*/ 
__global__ void find_max(int *rand, int numThreads) {
	
	int temp;
	int index = threadIdx.x + (blockDim.x * blockIdx.x);
		

	while(numThreads > WARP_SIZE){
		int halfway = numThreads / 2;	
		if (index < halfway){
			temp = rand[ index + halfway ];
			if (temp > rand[ index ]) {
				rand[index] = temp;
			}
		}
		__syncthreads();


		numThreads = halfway;	
	}
}

int main(int argc, char *argv[]){ 
	
	int *result = setup(argv[1]);
	int *devResult;
	int numBlocks;
	int m = 0;
	int i;
	
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int tPerBlock = prop.maxThreadsPerBlock;
	
	//Below condition only occcurs if nThreads is less than tPerBlock
	if ((nThreads % tPerBlock) != 0)
		numBlocks = (nThreads/tPerBlock) + 1;
	else
		numBlocks = nThreads/tPerBlock;
		
	
	hipMalloc((void**)&devResult, size * sizeof(int));
	
	hipMemcpy(devResult, result, size * sizeof(int), hipMemcpyHostToDevice);
	
	find_max<<<numBlocks, tPerBlock>>>(devResult, nThreads);
	
	hipMemcpy(result, devResult, size * sizeof(int), hipMemcpyDeviceToHost);
	

	for(i = 0; i < WARP_SIZE; i++) {
		if(result[i] > m) {
			m = result[i];
		}
	}
	
	printf("The max value in the array is: %d\n", m);
	
	
	//printf("Number of threads is: %d\n", nThreads);
	

	//printf("Num of blocks is: %d\n", numBlocks);
	//printf("%d\n", prop.maxThreadsPerBlock);
		
	
	free(result);
	hipFree(devResult);
	
	exit(0);
}
	
	
